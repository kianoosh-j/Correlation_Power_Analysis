#include "hip/hip_runtime.h"
#include<stdio.h>
#include <iostream>
#include<time.h>
#include<omp.h>
//#include "Dtools.h"
#include <time.h>
#include <sys/time.h>


typedef unsigned char BYTE;

#define length 40000
#define step 32
#define width 192
#define height 256
#define depth 16

#define test_byte 8

using namespace std;


__global__ void LargeScale_CorrCoef(int *a_dev, int *b_dev, float *c_dev){

		int tx = threadIdx.x, ty = threadIdx.y;
		int bx = blockIdx.x, by = blockIdx.y;

		int row = by*blockDim.y + ty;
		int col = bx*blockDim.x + tx;

		int exy = 0, ex = 0, ex2 = 0, ey = 0, ey2 = 0;
		float fexy = 0, fex = 0, fex2 = 0, fey = 0, fey2 = 0;

		//shared variables 
		__shared__ int a_shared[32][step];
		__shared__ int b_shared[step][16];
	//	__shared__ long long int btmp_shared[step][16];


		for (int it = 0 ; it < depth ; it++){
			for (int iter = 0 ; iter < length ; iter+=step){

				a_shared[ty][tx]    		  	  = a_dev[it*height*length + row*length + (tx+iter)];
				a_shared[ty][tx+blockDim.x]       = a_dev[it*height*length + row*length + (tx+blockDim.x+iter)];
				b_shared[ty][tx] 		  		  = b_dev[col + (ty+iter)*width];
				__syncthreads();


				for (int iter2 = 0 ; iter2 < step ; iter2++){
					ey  += b_shared[iter2][tx];
					ey2 += b_shared[iter2][tx]*b_shared[iter2][tx];
					ex  += a_shared[ty][iter2];
					ex2 += a_shared[ty][iter2]*a_shared[ty][iter2];
					exy += a_shared[ty][iter2]*b_shared[iter2][tx];
				}	
				__syncthreads();	

			}

			__syncthreads();

			
			fexy =  (float)((double)exy / (double)length);
			fex  =  (float)((double)ex  / (double)length);
			fex2 =  (float)((double)ex2 / (double)length);
			fey  =  (float)((double)ey  / (double)length);
			fey2 =  (float)((double)ey2 / (double)length);

			float cov  = fexy - fex*fey;
			float varx = fex2 - fex*fex;
			float vary = fey2 - fey*fey;
/*
		long long int cov  = length*exy - ex*ey;
		long long int varx = length*length*ex2 - length*ex*ex;
		long long int vary = length*length*ey2 - length*ey*ey;

*/		__syncthreads();


			c_dev[it*height*width + row*width + col] = (float)(cov/sqrt((float)(varx*vary)));
			ex = 0, ey = 0, ex2 = 0, ey2 = 0, exy = 0;
		}
}


void kernelCall(int *a, int *b, float *c){

struct timeval time1,time2,diff;
float sumOfTime = 0.0;
float sumTime=0.0;
for (int i = 0 ; i < 5 ; i++){
	dim3 Grid(12,8);
	dim3 Block(16,32);

	int *a_dev, *b_dev;
	float *c_dev;

	hipMalloc((void**)&a_dev, depth*height*length*sizeof(int) );
	hipMalloc((void**)&b_dev, width*length*sizeof(int) );
	hipMalloc((void**)&c_dev, depth*height*width*sizeof(float) );
	//hipMemset((void**)&c_dev, 0, depth*height*width*sizeof(float) );

	hipMemcpy(a_dev, a, depth*height*length*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, length*width*sizeof(int), hipMemcpyHostToDevice);
    cout<<"lunch"<<endl;
    gettimeofday(&time1,NULL);
    
	LargeScale_CorrCoef<<<Grid, Block>>>(a_dev, b_dev, c_dev);
	hipDeviceSynchronize();
	
	gettimeofday(&time2,NULL);
    timersub(&time2,&time1,&diff);
    hipMemcpy(c, c_dev, depth*height*width*sizeof(float), hipMemcpyDeviceToHost);
    
    sumOfTime = ((time2.tv_sec - time1.tv_sec)*1000.0)+((time2.tv_usec - time1.tv_usec)/1000.0);
    sumTime += sumOfTime;
    cout << "Elapsed time = " << (sumOfTime) << " ms" <<", Sec = "<< diff.tv_sec << ", ms = "<<diff.tv_usec/1000.0<<endl;
    
    //cout << "Elapsed Time = " << ((sumOfTime)) << " ms" <<endl;
    
	hipFree(a_dev);
	hipFree(b_dev);
	hipFree(c_dev);
	//hipDeviceReset();
}
	cout << "Avg. of total elapsed time = " << (sumTime)/5 << " ms" <<endl;
    //cout << "Elapsed Time = " << (double)((t2-t1)/1000.0) << " ms" <<endl;
    
}


void checkResult(int *a, int *b, float *c){
	long long int sum_b = 0, sum_a = 0.0, sum_b2 = 0.0, sum_a2 = 0.0, sum_ab = 0.0;
	int count = 0;
	float fsum_ab = 0.0, fsum_a = 0.0, fsum_a2 = 0.0, fsum_b = 0.0, fsum_b2 = 0.0;

	for (int iter = 0 ; iter < height ; iter++)
		for (int iter2 = 0 ; iter2 < width ; iter2++){
			for (int iter3 = 0 ; iter3 < length ; iter3++){
				sum_a  += a[test_byte*height*length + iter*length + iter3];
				sum_b  += b[iter3*width + iter2];
				sum_a2 += a[test_byte*height*length + iter*length + iter3]*a[test_byte*height*length + iter*length + iter3];
				sum_b2 += b[iter3*width + iter2]*b[iter3*width + iter2];
				sum_ab += a[test_byte*height*length + iter*length + iter3]*b[iter3*width + iter2];
			}


			fsum_ab = (float)((float)sum_ab / (float)length); //(float)
			fsum_a  = (float)((float)sum_a  / (float)length); //(float)
			fsum_a2 = (float)((float)sum_a2 / (float)length); //(float)
			fsum_b  = (float)((float)sum_b  / (float)length); //(float)
			fsum_b2 = (float)((float)sum_b2 / (float)length); //(float)

			float cov  = fsum_ab - fsum_a*fsum_b;
			float vara = fsum_a2 -fsum_a*fsum_a;
			float varb = fsum_b2 -fsum_b*fsum_b;

			float val_gpu = c[test_byte*height*width + iter*width + iter2];
			float val_cpu = /*cov/sqrt(vara*varb);*/(float)cov/sqrt(vara*varb);

			//if (val_cpu - val_gpu > 0.0000001910001 || val_gpu - val_cpu > 0.0000001910001)
			{
							count++;
//							if(iter == 0 && iter2 == 0)
							printf("c_dev[%d][%d] = %f, val_cpu = %f \n", iter, iter2, val_gpu, val_cpu);

			//	if ( iter == 255 && iter2 == 190)
			//		printf("c_dev[%d][%d] = %f, val_cpu = %f \n", iter, iter2, val_gpu, val_cpu);
//				if (count < height*width)
//					printf("c_dev[%d][%d] = %f, val_cpu = %f \n", iter, iter2, val_gpu, val_cpu);
			}
				
			sum_a  = 0.0;
			sum_a2 = 0.0;
			sum_b  = 0.0;
			sum_b2 = 0.0;
			sum_ab = 0.0;
		}

		printf(" error rate == %d \n", count);
		//printf("c[0] = %f\n", c[0]);
}

int main(){

        int   *a  = new int[depth*height*length];
		int   *b  = new int[length*width];
        float *c  = new float[depth*width*height];

    int temp;
    srand(time(NULL));

    for (int it = 0 ; it < depth ; it++){
    	for (int iter = 0 ; iter < height ; iter++)
			for (int iter2 = 0 ; iter2 < length ; iter2++){
				temp = rand()%10;
				a[it*height*length + iter*length + iter2] = (int)temp;
			}		
    }
	
	for (int iter = 0 ; iter < length ; iter++)		
		for (int iter2 = 0 ; iter2 < width ; iter2++){
			temp = rand()%10;	
			b[iter*width + iter2] = (int)temp;	
		}

	kernelCall(a,b,c);
	cout<<"Checking Result"<<endl;
	checkResult(a,b,c);	

	return 0;
}
